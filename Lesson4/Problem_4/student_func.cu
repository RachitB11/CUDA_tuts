#include "hip/hip_runtime.h"
//Udacity HW 4
//Radix Sorting

#include "utils.h"
#include <thrust/host_vector.h>
#include<bitset>
#define RADIX_NUMBER 1 // This is the log of number of radixbits
/* Red Eye Removal
   ===============

   For this assignment we are implementing red eye removal.  This is
   accomplished by first creating a score for every pixel that tells us how
   likely it is to be a red eye pixel.  We have already done this for you - you
   are receiving the scores and need to sort them in ascending order so that we
   know which pixels to alter to remove the red eye.

   Note: ascending order == smallest to largest

   Each score is associated with a position, when you sort the scores, you must
   also move the positions accordingly.

   Implementing Parallel Radix Sort with CUDA
   =============================
   =============

   The basic idea is to construct a histogram on each pass of how many of each
   "digit" there are.   Then we scan this histogram so that we know where to put
   the output of each digit.  For example, the first 1 must come after all the
   0s so we have to know how many 0s there are to be able to start moving 1s
   into the correct position.

   1) Histogram of the number of occurrences of each digit
   2) Exclusive Prefix Sum of Histogram
   3) Determine relative offset of each digit
        For example [0 0 1 1 0 0 1]
                ->  [0 1 0 1 2 3 2]
   4) Combine the results of steps 2 & 3 to determine the final
      output location for each element and move it there

   LSB Radix sort is an out-of-place sort and you will need to ping-pong values
   between the input and output buffers we have provided.  Make sure the final
   sorted results end up in the output buffer!  Hint: You may need to do a copy
   at the end.
 */

  // Aggregate the output data using all the information calculated
  void __global__ generateOutput(unsigned int* const d_outputVals,
    unsigned int* const d_outputPos, const unsigned int* const d_inputVals,
     const unsigned int* const d_inputPos,
     const unsigned int* const d_relative_offsets,
     const unsigned int* const d_local_cdf, unsigned int* const d_cdf,
     const unsigned int* const d_predicate, const size_t numBins,
     const size_t numElems)
   {
     int tid = threadIdx.x;
     int bid = blockIdx.x;
     int myId = tid + blockDim.x * bid;

     if(myId<numElems)
     {
       int predicate = d_predicate[myId];

       int in_grid_position = d_relative_offsets[myId];
       int in_bin_position = d_local_cdf[bid*numBins + predicate] + in_grid_position;
       int global_position = d_cdf[predicate] + in_bin_position;

       if(global_position<numElems)
       {
         d_outputVals[global_position] = d_inputVals[myId];
         d_outputPos[global_position] = d_inputPos[myId];
       }
     }
   }

  // Generate the relative offsets of each set using a compact and segmented scan
  // Note that this is within a single thread block
  void __global__ generateScanRelativeOffsets(unsigned int* const d_relative_offsets,
    const unsigned int* const d_compact, const size_t numElems,
    const size_t numBins)
  {
    // soffset data is numBins x blockDim.x
    extern __shared__ unsigned int sscandata[];

    unsigned int tid = threadIdx.x;
    unsigned int bid = blockIdx.x;
    unsigned int grid = bid/numBins;
    unsigned int bin = bid%numBins;
    unsigned int linearId = tid + grid*blockDim.x;

    unsigned int n = blockDim.x;
    unsigned int pout = 0;
    unsigned int pin = 1;


    unsigned int compact_val = 0;

    if(linearId<numElems)
      compact_val = d_compact[numElems*bin + linearId];

    sscandata[pout*n+tid] = compact_val;
    __syncthreads();

    for (unsigned int s = 1; s < blockDim.x ; s <<= 1)
    {
      // Swap pin and pout
      pout = 1 - pout;
      pin = 1 - pout;
      if(linearId<numElems)
      {
        // Do not touch if the data has no neighbour s indexes left to it.
        if(tid>=s)
        {
          sscandata[pout*n+tid] = sscandata[pin*n+tid] + sscandata[pin*n+tid-s];
        }
        else
        {
          sscandata[pout*n+tid] = sscandata[pin*n+tid];
        }
      }
      __syncthreads();        // make sure all adds at one stage are done!
    }
    if(linearId<numElems && compact_val==1)
    {
      // You need to subtract by -1 to ensure exclusive scan
      d_relative_offsets[linearId] = sscandata[pout*n+tid]-1;
    }
  }

  // Generate the compact using the predicate
  void __global__ generateCompact(unsigned int* const d_compact,
    const unsigned int* const d_predicate, const unsigned int numBins,
    const unsigned int numElems)
  {
    unsigned int tid = threadIdx.x;
    unsigned int bid = blockIdx.x;
    // unsigned int myId = tid + bid*blockDim.x;
    unsigned int grid = bid/numBins;
    unsigned int bin = bid%numBins;

    unsigned int linearId = tid + grid*blockDim.x;

    if(linearId<numElems)
      d_compact[numElems*bin+linearId] = (d_predicate[linearId]==bin) ? 1:0;
  }

  // Generate the global cdf
  void __global__ generateGlobalCdf(unsigned int* const d_cdf,
    const unsigned int* const d_histogram)
  {
    extern __shared__ float sglobalhisto[]; // allocated on invocation

    int n = blockDim.x;
    int bin = threadIdx.x;
    int pout = 0, pin = 1;

    // This is exclusive scan, so shift right by one and set first element to 0
    sglobalhisto[pout*n + bin] = (bin > 0) ? d_histogram[bin-1] : 0;
    __syncthreads();

    for (int s = 1; s < blockDim.x; s *= 2)
    {
      pout = 1 - pout; // swap double buffer indices
      pin = 1 - pout;
      if (bin >= s)
        sglobalhisto[pout*n+bin] = sglobalhisto[pin*n+bin] + sglobalhisto[pin*n+bin-s];
      else
        sglobalhisto[pout*n+bin] = sglobalhisto[pin*n+bin];
     __syncthreads();
    }
    d_cdf[bin] = sglobalhisto[pout*n+bin]; // write output
  }

  // Generate the global histogram
  void __global__ generateGlobalHistogram( unsigned int* const d_histogram,
    const unsigned int* const d_local_cdf, const unsigned int* const d_local_histogram,
    const size_t gridSize)
  {
    int bin = threadIdx.x;
    int idx_final_bin = blockDim.x*(gridSize-1) + bin;
    d_histogram[bin] = d_local_cdf[idx_final_bin] + d_local_histogram[idx_final_bin];
  }

  // Generate the local cdf of the histograms along the grid. Here the d_histogram is of size
  // gridSize x numBins
  void __global__ generateLocalCdf( unsigned int* const d_local_cdf,
    const unsigned int* const d_local_histogram, const size_t numBins)
  {
    extern __shared__ float shistodata[]; // allocated on invocation

    // Thread id
    int grid = threadIdx.x;
    int bin = blockIdx.x;

    // Tells me which part is storing the in and out at the current step
    int pout = 0, pin = 1;

    // Store the total number of elements
    int n = blockDim.x;

    // Copy the data to the shared memory
    // shistodata[tid] = d_local_histogram[tid];
    // If we're doing an exclusive scan copy the data from n-1 and set position
    // 0 to identity which in this case is 0
    shistodata[pout*n + grid] = (grid>0) ? d_local_histogram[(grid-1)*numBins + bin] : 0;
    __syncthreads();

    // Do the hillis steele reduction along the grid dimension only
    for (int s = 1; s < blockDim.x; s *= 2)
    {
      pout = 1 - pout; // swap double buffer indices
      pin = 1 - pout;

      if (grid >= s)
          shistodata[pout*n + grid] = shistodata[pin*n + grid] + shistodata[pin*n + grid-s];
      else
          shistodata[pout*n + grid] = shistodata[pin*n + grid];
      __syncthreads();
    }

    // Populate the grid row of the cdf
    d_local_cdf[grid*numBins + bin] = shistodata[pout*n + grid];
  }

  // Generate local histograms and populate the predicate
  void __global__ generateLocalHistograms(unsigned int* const d_local_histogram,
    unsigned int* const d_predicate, const unsigned int* const d_inputVals,
    const size_t numElems, const size_t numBins, const unsigned int mask,
    const unsigned int shift)
  {
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int bid = blockIdx.x;

    if(myId<numElems)
    {
      unsigned int val = d_inputVals[myId];

      // Find the digit by applying the mask
      unsigned int digit = (val&mask)>>shift;

      // Update the predicate
      d_predicate[myId] = digit;

      // Add the point to the local histogram at position (grid,digit)
      atomicAdd(&(d_local_histogram[bid*numBins + digit]), 1);
    }
  }

  // The main radix step
  void radixSortStep(const unsigned int* const d_inputVals,
    const unsigned int* const d_inputPos, unsigned int* const d_outputVals,
    unsigned int* const d_outputPos, unsigned int* const d_local_histogram,
    unsigned int* const d_local_cdf, unsigned int* const d_histogram,
    unsigned int* const d_cdf, unsigned int* const d_predicate,
    unsigned int* const d_compact, unsigned int* const d_relative_offsets,
    const size_t blockSize, const size_t gridSize, const size_t numBins,
    const size_t numElems, const unsigned int radixBits, const int place,
    const unsigned int seed_mask)
  {
//////////////Define the shift while evaluating this place//////////////////////
    unsigned int shift = place*radixBits;
    unsigned int mask = seed_mask<<shift;

    // NOTE: Debug
    // std::cout<<"Mask "<<place<<" : "<<std::bitset<sizeof(unsigned int)*8>(mask)<<std::endl;
    // std::cout<<"numBins "<<place<<" : "<<numBins<<std::endl;
    // std::cout<<"numElems "<<place<<" : "<<numElems<<std::endl;

//////Populate the local histogram (gridSize x numBins) and the predicates//////
    generateLocalHistograms<<<gridSize, blockSize>>>(d_local_histogram,
      d_predicate, d_inputVals, numElems, numBins, mask, shift);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    // NOTE: Debug
    // unsigned int h_local_histogram[gridSize*numBins];
    // unsigned int sum=0;
    // checkCudaErrors(hipMemcpy(h_local_histogram, d_local_histogram, gridSize*numBins*sizeof(unsigned int), hipMemcpyDeviceToHost));
    // for(size_t i=0;i<gridSize*numBins;i++)
    // {
    //   if(h_local_histogram[i]>1024)
    //     std::cout<<h_local_histogram[i]<<" ";
    //   sum+=h_local_histogram[i];
    // }
    // std::cout<<sum<<std::endl;

///// Populate the local cdf (gridSize x numBins)along the grid dimension //////
    // It is this local cdf that enables connecting the scans in individual
    // thread blocks
    generateLocalCdf<<<numBins, gridSize, 2*gridSize*sizeof(unsigned int)>>>(
      d_local_cdf, d_local_histogram, numBins);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    // NOTE: Debug
    // unsigned int h_local_cdf[gridSize*numBins];
    // unsigned int sum=0;
    // checkCudaErrors(hipMemcpy(h_local_cdf, d_local_cdf, gridSize*numBins*sizeof(unsigned int), hipMemcpyDeviceToHost));
    // for(size_t i=(gridSize-1)*numBins;i<gridSize*numBins;i++)
    // {
    //     sum+=(h_local_cdf[i]+h_local_histogram[i]);
    // }
    // std::cout<<sum<<std::endl;

///////////////// Populate the global histogram (numBins) //////////////////////
    generateGlobalHistogram<<<1, numBins>>>(d_histogram, d_local_cdf, d_local_histogram,
      gridSize);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    //NOTE: Debug
    // unsigned int h_histogram[numBins];
    // int sum = 0;
    // checkCudaErrors(hipMemcpy(h_histogram, d_histogram, numBins*sizeof(unsigned int), hipMemcpyDeviceToHost));
    // for(size_t i=0;i<numBins;i++)
    // {
    //   // std::cout<<h_histogram[i]<<" ";
    //   sum+=h_histogram[i];
    // }
    // std::cout<<sum<<std::endl;

//////////////// Populate the global cdf (gridSize)/////////////////////////////
    generateGlobalCdf<<<1, numBins, 2*numBins*sizeof(unsigned int)>>>(d_cdf,
      d_histogram);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    //NOTE: Debug
    // unsigned int h_cdf[numBins];
    // checkCudaErrors(hipMemcpy(h_cdf, d_cdf, numBins*sizeof(unsigned int), hipMemcpyDeviceToHost));
    // for(size_t i=0;i<numBins;i++)
    // {
    //   std::cout<<h_cdf[i]<<" ";
    // }
    // std::cout<<std::endl;
    // std::cout<<h_cdf[numBins-1]+h_histogram[numBins-1]<<std::endl;

////////////////// Generate the compact (numBins x numElems) list///////////////
    // Note that the size of d_compact is numBins x numElems where each row
    // represents the compact of each element for a particular bin/digit
    generateCompact<<<gridSize*numBins, blockSize>>>(d_compact, d_predicate,
      numBins, numElems);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    // NOTE: Debug
    // unsigned int* h_compact = new unsigned int[numBins*numElems];
    // checkCudaErrors(hipMemcpy(h_compact, d_compact, numBins*numElems*sizeof(unsigned int), hipMemcpyDeviceToHost));
    // unsigned int sum=0;
    // for(size_t i=0;i<numBins*numElems;i++)
    // {
    //   sum+=h_compact[i];
    // }
    // std::cout<<sum<<std::endl;
    // delete h_compact;

/////////////// Estimate the relative offsets (numElems) in each block /////////
    //Note that the relative offsets are the offsets of the digit of the element
    // in that "place" in the block for that digit. You need to get the offset
    // between blocks for each digit using the local cdf. You need to get the
    // global offset between digits using the global cdf.
    // So:
    // relative_offset:  Facilitates within block and digit position estimation
    // local_cdf: Facilitates within digit position estimation
    // global_cdf: Facilitates total position estimation
    generateScanRelativeOffsets<<<gridSize*numBins, blockSize,
      2*blockSize*sizeof(unsigned int)>>>(d_relative_offsets, d_compact, numElems,
        numBins);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());


///////////// Estimate the final position of the block and move the data///////
    // 1. Use the relative offset for offset within a block
    // 2. Use the local_cdf for grid based offset for offset within a bin
    // 3. Use the global cdf to compute the offset in the entire list
    // Use the predicate to access the correct bin in each of the above cases.
    generateOutput<<<gridSize, blockSize>>>(d_outputVals, d_outputPos, d_inputVals,
      d_inputPos, d_relative_offsets, d_local_cdf, d_cdf, d_predicate, numBins,
      numElems);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

///////////////////////////////////END//////////////////////////////////////////
  }


void your_sort(unsigned int* const d_inputVals,
               unsigned int* const d_inputPos,
               unsigned int* const d_outputVals,
               unsigned int* const d_outputPos,
               const size_t numElems)
{
  //TODO
  //PUT YOUR SORT HERE
  // So a radix sort is you take the LSB move all the 0s to the top and all the 1s
  // to the bottom

  // Use powers of 2 as the radix bits
  // This is important so that the numPlaces are even
  const unsigned int radixBits = pow(2,RADIX_NUMBER);
  const unsigned int seed_mask = pow(2,radixBits)-1;
  const size_t numBins = pow(2,radixBits);

  // Assuming unsigned int size of 32 number of places equals 32/radixBits
  const size_t numPlaces = sizeof(unsigned int)*8/radixBits;
  int binBytes = numBins * sizeof(unsigned int);
  int elemBytes = numElems * sizeof(unsigned int);
  int allBytes = numBins * numElems * sizeof(unsigned int);

  // Assign the block and the grid size
  const size_t blockSize = 512;
  const size_t gridSize = numElems/blockSize + 1;

  // Contains the local per grid histogram information
  // Has size equals gridSize x numBins. It contains the histogram per grid
  unsigned int* d_local_histogram;
  checkCudaErrors(hipMalloc((void**) &d_local_histogram, gridSize*binBytes));
  checkCudaErrors(hipMemset((void *) d_local_histogram, 0, gridSize*binBytes));

  // Contains the local per grid cdf information cdf along the grid dimension
  // Has size equals gridSize x numBins. It contains the cdf in the grid dimension
  unsigned int* d_local_cdf;
  checkCudaErrors(hipMalloc((void**) &d_local_cdf, gridSize*binBytes));
  checkCudaErrors(hipMemset((void *) d_local_cdf, 0, gridSize*binBytes));

  // Contains the global histogram information
  // Has size equals numBins representing all possible digits in a place
  unsigned int* d_histogram;
  checkCudaErrors(hipMalloc((void**) &d_histogram, binBytes));
  checkCudaErrors(hipMemset((void *) d_histogram, 0, binBytes));

  // Contains the global cdf per bin
  // Has size equals numBins representing all possible digits in a place
  unsigned int* d_cdf;
  checkCudaErrors(hipMalloc((void**) &d_cdf, binBytes));
  checkCudaErrors(hipMemset((void *) d_cdf, 0, binBytes));

  // This contains the digit at a place information for each element
  // Has size equals numElems
  unsigned int* d_predicate;
  checkCudaErrors(hipMalloc((void**) &d_predicate, elemBytes));
  checkCudaErrors(hipMemset((void *) d_predicate, 0, elemBytes));

  // This contains the relative offsets of each element
  // Has size equals numElems
  unsigned int* d_relative_offsets;
  checkCudaErrors(hipMalloc((void**) &d_relative_offsets, elemBytes));
  checkCudaErrors(hipMemset((void *) d_relative_offsets, 0, elemBytes));

  // Make an array to store the compact list for each bin
  // Has size equal numBins x numElems
  unsigned int* d_compact;
  checkCudaErrors(hipMalloc((void**) &d_compact, allBytes));
  checkCudaErrors(hipMemset((void *) d_compact, 0, allBytes));

  // I'm guaranteed to have even number of places with the way the RADIX NUMBER
  // and radix digits are defined. So I don't need to check whether the output
  // arrays actually contain the final output.
  for(unsigned int i=0; i<numPlaces; i++)
  {
    // Do a step of the radix Sort, swap input and output each step
    if(i%2==0)
      radixSortStep(d_inputVals, d_inputPos, d_outputVals, d_outputPos,
        d_local_histogram, d_local_cdf, d_histogram, d_cdf, d_predicate,
        d_compact, d_relative_offsets, blockSize, gridSize, numBins, numElems, radixBits,
        i, seed_mask);
    else
      radixSortStep(d_outputVals, d_outputPos, d_inputVals, d_inputPos,
        d_local_histogram, d_local_cdf, d_histogram, d_cdf, d_predicate,
        d_compact, d_relative_offsets, blockSize, gridSize, numBins, numElems,
        radixBits, i, seed_mask);

    // Reset all the data to 0 for the next step
    checkCudaErrors(hipMemset((void *) d_local_histogram, 0, gridSize*binBytes));
    checkCudaErrors(hipMemset((void *) d_local_cdf, 0, gridSize*binBytes));
    checkCudaErrors(hipMemset((void *) d_histogram, 0, binBytes));
    checkCudaErrors(hipMemset((void *) d_cdf, 0, binBytes));
    checkCudaErrors(hipMemset((void *) d_predicate, 0, elemBytes));
    checkCudaErrors(hipMemset((void *) d_relative_offsets, 0, elemBytes));
    checkCudaErrors(hipMemset((void *) d_compact, 0, allBytes));
  }

  // Note that the numPlaces are even so the final ouput should be in the output
  // fields

  // Free all the assigned arrays
  checkCudaErrors(hipFree(d_compact));
  checkCudaErrors(hipFree(d_relative_offsets));
  checkCudaErrors(hipFree(d_predicate));
  checkCudaErrors(hipFree(d_cdf));
  checkCudaErrors(hipFree(d_histogram));
  checkCudaErrors(hipFree(d_local_cdf));
  checkCudaErrors(hipFree(d_local_histogram));

}
