#include "hip/hip_runtime.h"
#include <stdio.h>
#include "gputimer.h"

#define NUM_THREADS 1000000
#define ARRAY_SIZE  100

#define BLOCK_WIDTH 1000

void print_array(int *array, int size)
{
    printf("{ ");
    for (int i = 0; i < size; i++)  { printf("%d ", array[i]); }
    printf("}\n");
}

__global__ void increment_naive(int *g)
{
	// which thread is this?
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	// each thread to increment consecutive elements, wrapping at ARRAY_SIZE
	i = i % ARRAY_SIZE;
  // This should lead to each element in the array having NUM_THREADS/ARRAY_SIZE in the arrays
  // But here we will get a race condition a condition where multiple threads are
  // trying to access the same memory.
	g[i] = g[i] + 1;
}

__global__ void increment_atomic(int *g)
{
	// which thread is this?
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	// each thread to increment consecutive elements, wrapping at ARRAY_SIZE
	i = i % ARRAY_SIZE;
  // This should lead to each element in the array having NUM_THREADS/ARRAY_SIZE in the arrays
	atomicAdd(& g[i], 1);
}

int main(int argc,char **argv)
{
    GpuTimer timer;
    printf("%d total threads in %d blocks writing into %d array elements\n",
           NUM_THREADS, NUM_THREADS / BLOCK_WIDTH, ARRAY_SIZE);

    // declare and allocate host memory
    int h_array[ARRAY_SIZE];
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(int);

    // declare, allocate, and zero out GPU memory
    int * d_array;
    hipMalloc((void **) &d_array, ARRAY_BYTES);
    hipMemset((void *) d_array, 0, ARRAY_BYTES);

    // launch the kernel - comment out one of these
    timer.Start();

    // Here there will be a race condition since many threads are trying to update
    // the same memory location. This will lead to it getting values lower than
    // NUM_THREADS/ARRAY_SIZE
    // increment_naive<<<NUM_THREADS/BLOCK_WIDTH, BLOCK_WIDTH>>>(d_array);

    // Here by atomicizing the global array we have prevented the possibility of
    // a race condition and so every block should have value of NUM_THREADS/ARRAY_SIZE
    // using this kernel
    increment_atomic<<<NUM_THREADS/BLOCK_WIDTH, BLOCK_WIDTH>>>(d_array);
    timer.Stop();

    // copy back the array of sums from GPU and print
    hipMemcpy(h_array, d_array, ARRAY_BYTES, hipMemcpyDeviceToHost);
    print_array(h_array, ARRAY_SIZE);
    printf("Time elapsed = %g ms\n", timer.Elapsed());

    // free GPU memory allocation and exit
    hipFree(d_array);
    return 0;
}
