
#include <hip/hip_runtime.h>
#include <stdio.h>

class CudaClass
{
public:
int* data;
CudaClass(int x) {
    data = new int[1]; data[0] = x;
}
};

__global__ void useClass(CudaClass *cudaClass)
{
    printf("%d\n", cudaClass->data[0]);
};




int main()
{
    CudaClass c(1);
    // create class storage on device and copy top level class
    CudaClass *d_c;
    hipMalloc((void **)&d_c, sizeof(CudaClass));
    hipMemcpy(d_c, &c, sizeof(CudaClass), hipMemcpyHostToDevice);
    // make an allocated region on device for use by pointer in class
    int *hostdata;
    hipMalloc((void **)&hostdata, sizeof(int));
    hipMemcpy(hostdata, c.data, sizeof(int), hipMemcpyHostToDevice);
    // copy pointer to allocated device storage to device class
    hipMemcpy(&(d_c->data), &hostdata, sizeof(int *), hipMemcpyHostToDevice);
    useClass<<<1,1>>>(d_c);
    hipDeviceSynchronize();
    return 0;
}
